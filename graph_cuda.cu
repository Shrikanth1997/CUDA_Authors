#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// Define maximum number of vertices in the graph
#define N 317080
#define EDGES 1049886

int auth_num[N+1];
// Data structure to store graph
struct Graph {
	// An array of pointers to Node to represent adjacency list
	struct Node* head[N+1];
};

// A data structure to store adjacency list nodes of the graph
struct Node {
	int dest;
	struct Node* next;
};

// data structure to store graph edges
struct Edge {
	int src, dest;
};

extern __managed__ struct Graph * graph ;
extern __managed__ struct Node* newNode ;

// Function to create an adjacency list from specified edges
__host__ void createGraph(struct Edge edges[], int n)
{
	unsigned i;

	// allocate memory for graph data structure
	struct Graph* graph = (struct Graph*)malloc(sizeof(struct Graph));

	// initialize head pointer for all vertices
	for (i = 0; i < N+1; i++){
		graph->head[i] = NULL;

	}

	// add edges to the directed graph one by one
	for (i = 0; i < n; i++)
	{
		// get source and destination vertex
		int src = edges[i].src;
		int dest = edges[i].dest;

		// allocate new node of Adjacency List from src to dest
		hipMallocManaged(&newNode, sizeof(struct Node), (unsigned int)hipMemAttachGlobal);
       		hipMemAdvise(newNode, sizeof(struct Node), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

		//struct Node* newNode = (struct Node*)malloc(sizeof(struct Node));
		newNode->dest = dest;

		// point new node to current head
		newNode->next = graph->head[src];

		// point head pointer to new node
		graph->head[src] = newNode;


		// 2. allocate new node of Adjacency List from dest to src
		hipMallocManaged(&newNode, sizeof(struct Node), (unsigned int)hipMemAttachGlobal);
       		hipMemAdvise(newNode, sizeof(struct Node), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

		//newNode = (struct Node*)malloc(sizeof(struct Node));
		newNode->dest = src;

		// point new node to current head
		newNode->next = graph->head[dest];
	
		// change head pointer to point to the new node
		graph->head[dest] = newNode;
	}

	//return graph;
}

// Function to print adjacency list representation of graph
__global__ void countAuth(struct Graph* graph,int auth_num[], int n)
{
    	int tid = blockIdx.x * blockDim.x + threadIdx.x; // HERE


	//int i;
	//for (i = 0; i < n+1; i++)
	//{
		int co_auth = 0;
		// print current vertex and all ts neighbors
		struct Node* ptr = graph->head[tid];
		while (ptr != NULL)
		{
			//printf("(%d -> %d)\t", tid, ptr->dest);
			ptr = ptr->next;
			co_auth++;
		}
		auth_num[tid] = co_auth;
		//auth_num[tid] = ptr->dest;
		//printf("\n");
	//}
}

long get_vert(char *str){
	char vert[20];
	int space_count = 0;
	int num_vert=0;	
	
	int i=0, j=0;
	while(str[i] != '\n'){
	
		if(str[i] == ' ')
			space_count++;
		if(space_count == 2){
			vert[j] = str[i];
			j++;
		}
		else if(space_count>2)	
			break;
		i++;
	}
	vert[j] = '\0';
    	//printf("%s\n", vert);
	num_vert = atoi(vert);
    	//printf("%d\n", num_vert);
	return num_vert;
	
}

int get_src(char *str){
	char s[20];
        int space_count = 0;
        int src=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 0){
                        s[j] = str[i];
                        j++;
                }
		else
			break;
                i++;
        }
        s[j] = '\0';
        //printf("%s\n", s);
        src = atoi(s);
        //printf("%d\n", src);
        return src;
}

int get_dst(char *str){
	char d[20];
        int space_count = 0;
        int dst=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 1){
                        d[j] = str[i];
                        j++;
                }
		else if(space_count>1)
			break;
                i++;
        }
        d[j] = '\0';
        //printf("%s\n", d);
        dst = atoi(d);
        //printf("%d\n", dst);
        return dst;
}

// Directed Graph Implementation in C
int main(void)
{
	// input array containing edges of the graph (as per above diagram)
	// (x, y) pair in the array represents an edge from x to y
	struct Edge *edges;
	edges = (struct Edge *) calloc (EDGES, sizeof(struct Edge));

    FILE *fp;
    char str[200];
    const char* file = "dblp-co-authors.txt";
    //const char* file = "test.txt";
 
    fp = fopen(file, "r");
    if (fp == NULL){
        printf("Could not open file %s",file);
        return 1;
    }
    
	int vert, i=0;
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    //printf("%s", str);
	    vert = get_vert(str);
	    long src, dst;
	    //new_graph(vert);
	//struct graph* gph = new_graph(vert); 
	    while (fgets(str, 200, fp) != NULL){
		//printf("%s", str);
		src = get_src(str);
		dst = get_dst(str);
		edges[i].src = src;
		edges[i].dest = dst;
		i++;
	    }
	
	printf("Edges copied....\n");

	// calculate number of edges
	int n = sizeof(edges)/sizeof(edges[0]);

	hipMallocManaged(&graph, sizeof(struct Graph), (unsigned int)hipMemAttachGlobal);
    	hipMemAdvise(graph, sizeof(struct Graph), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

	createGraph(edges, N);

	printf("Graph Created...\n");	


	int graph_size = N+1;
    	int block_size  = 512;
    	int grid_size   = ((graph_size-1)/block_size) + 1;

    	// Set device that we will use for our cuda code
    	hipSetDevice(0);
	
	for (i = 0; i < N+1; i++){
		auth_num[i] = 0;

	}

	int *auth_num_gpu;
	hipMalloc(&auth_num_gpu, (N+1)*sizeof(int));
	hipMemcpy(auth_num_gpu, auth_num, N+1, hipMemcpyHostToDevice);

	// print adjacency list representation of graph
	countAuth<<<N+1, 1>>>(graph,auth_num_gpu, N);

	hipMemcpy(auth_num, auth_num_gpu, N+1, hipMemcpyDeviceToHost);

	for(i=0;i<N+1;i++){
		printf("Author %d : %d\n",i,auth_num[i]);
	}
	
	//hipFree(auth_num_gpu);

	/*for ( int i=0; i<= N; ++i){
		hipFree(graph->head[i]);
	}*/
	//hipFree(graph);

	return 0;
}
