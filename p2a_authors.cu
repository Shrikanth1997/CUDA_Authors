#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

struct node{
	int dst;
	struct node* next;
};

struct list{
	struct node *head;
};

struct graph{
	int n;
	struct list* set;
};



extern __managed__ struct node* newnode;
extern __managed__ struct graph* newgraph;

/*struct node* new_node(int dst){
	hipMallocManaged(&newnode, sizeof(struct node), (unsigned int)hipMemAttachGlobal);
	hipMemAdvise(newnode, sizeof(struct node), hipMemAdviseSetAccessedBy, hipCpuDeviceId);
	newnode -> dst = dst;
	newnode -> next = NULL;

	return newnode;
}*/

void new_node(int dst){
	hipMallocManaged(&newnode, sizeof(struct node), (unsigned int)hipMemAttachGlobal);
	hipMemAdvise(newnode, sizeof(struct node), hipMemAdviseSetAccessedBy, hipCpuDeviceId);
	newnode -> dst = dst;
	newnode -> next = NULL;

}

/*struct graph* new_graph(int n){
	
	hipMallocManaged(&newgraph, sizeof(struct graph), (unsigned int)hipMemAttachGlobal);
	hipMemAdvise(newgraph, sizeof(struct graph), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

	newgraph -> n = n;
	
	newgraph -> set = (struct list*)malloc(n * sizeof(struct list)) ;

	int i;
	for(i=0;i<n;i++)
		newgraph->set[i].head = NULL;

	return newgraph;

}*/

void new_graph(int n){
	
	hipMallocManaged(&newgraph, sizeof(struct graph), (unsigned int)hipMemAttachGlobal);
	hipMemAdvise(newgraph, sizeof(struct graph), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

	newgraph -> n = n;
	
	newgraph -> set = (struct list*)malloc(n * sizeof(struct list)) ;

	int i;
	for(i=0;i<n;i++)
		newgraph->set[i].head = NULL;


}

/*void addEdge(struct graph* gph, int src, int dst){
	struct node* newnode = new_node(dst);
	newnode->next = gph->set[src].head;
	gph->set[src].head = newnode;

	newnode = new_node(src);
        newnode->next = gph->set[dst].head;
        gph->set[dst].head = newnode;
}*/

void addEdge( int src, int dst){
	new_node(dst);
	newnode->next = newgraph->set[src].head;
	newgraph->set[src].head = newnode;

	new_node(src);
        newnode->next = newgraph->set[dst].head;
        newgraph->set[dst].head = newnode;
}


__global__ void count(int* auth_num) {
    
    // Calculate the index in the vector for the thread using the internal variables
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // HERE
    
    // This if statement is added in case we have more threads executing
    // Than number of elements in the vectors. How can this help?

        int co_auth = 0; 
        struct node* vert_node = newgraph->set[tid].head; 
        //printf("\n Adjacency list of vertex %d\n head ", v); 
        /*while (vert_node) 
        { 
            //printf("-> %d", vert_node->dst); 
            vert_node = vert_node->next;
	    co_auth++; 
        }*/
        auth_num[tid] = vert_node->dst;
    

}


//Utility functions to read the file
long get_vert(char *str){
	char vert[20];
	int space_count = 0;
	int num_vert=0;	
	
	int i=0, j=0;
	while(str[i] != '\n'){
	
		if(str[i] == ' ')
			space_count++;
		if(space_count == 2){
			vert[j] = str[i];
			j++;
		}
		else if(space_count>2)	
			break;
		i++;
	}
	vert[j] = '\0';
    	//printf("%s\n", vert);
	num_vert = atoi(vert);
    	//printf("%d\n", num_vert);
	return num_vert;
	
}

int get_src(char *str){
	char s[20];
        int space_count = 0;
        int src=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 0){
                        s[j] = str[i];
                        j++;
                }
		else
			break;
                i++;
        }
        s[j] = '\0';
        //printf("%s\n", s);
        src = atoi(s);
        //printf("%d\n", src);
        return src;
}

int get_dst(char *str){
	char d[20];
        int space_count = 0;
        int dst=0;

        int i=0, j=0;
        while(str[i] != '\n'){

                if(str[i] == ' ')
                        space_count++;
                if(space_count == 1){
                        d[j] = str[i];
                        j++;
                }
		else if(space_count>1)
			break;
                i++;
        }
        d[j] = '\0';
        //printf("%s\n", d);
        dst = atoi(d);
        //printf("%d\n", dst);
        return dst;
}

int compare (const void * a, const void * b)
{
  return ( *(int*)b - *(int*)a );
}


int main() { 

    FILE *fp;
    char str[200];
    const char* file = "dblp-co-authors.txt";
 
    fp = fopen(file, "r");
    if (fp == NULL){
        printf("Could not open file %s",file);
        return 1;
    }
    
	int vert;
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    fgets(str, 200, fp);
	    //printf("%s", str);
	    vert = get_vert(str);
	    long src, dst;
	    new_graph(vert);
	//struct graph* gph = new_graph(vert); 
	    while (fgets(str, 200, fp) != NULL){
		//printf("%s", str);
		src = get_src(str);
		dst = get_dst(str);
		addEdge(src,dst);
	    }
   

    printf("Graph Created....\n");

    
    /*for(int v=0;v<10;v++){
        struct node* vert_node = newgraph->set[v].head; 
	checkauth=0;
        printf("\n Adjacency list of vertex %d\n head ", v); 
        while (vert_node) 
        { 
            printf("-> %d", vert_node->dst); 
            vert_node = vert_node->next;
        }
     }*/


    // Set GPU Variables based on input arguments
    int graph_size = newgraph->n;
    int block_size  = 512;
    int grid_size   = ((graph_size-1)/block_size) + 1;

    // Set device that we will use for our cuda code
    hipSetDevice(0);
        
    // Time Variables
    hipEvent_t start, stop;
    float time;
    hipEventCreate (&start);
    hipEventCreate (&stop);

    // Input Arrays and variables
    int *auth_num    = new int [graph_size];

    // Pointers in GPU memory
    int *auth_num_gpu;
    struct graph *gph_gpu;

    int actual_size = 1049866  * sizeof(struct graph);
    int num_size = graph_size * sizeof(int);

    // allocate the memory on the GPU
    //hipMalloc(&gph_gpu, actual_size);
    //hipMalloc(&auth_num_gpu, num_size);

    // copy the arrays 'a' and 'b' to the GPU
    //hipMemcpy(gph_gpu,gph,actual_size,hipMemcpyHostToDevice);

    //
    // GPU Calculation
    ////////////////////////

    printf("Counting....\n");

    hipEventRecord(start,0);

    // call the kernel
    //count<<<grid_size,block_size>>>(auth_num_gpu);
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    //printf("\tParallel Job Time: %.2f ms\n", time);

    // copy the array 'c' back from the GPU to the CPU
    // HERE (there's one more at the end, don't miss it!)
    //hipMemcpy(auth_num,auth_num_gpu,num_size,hipMemcpyDeviceToHost);
    
    /*for(int i=0;i<graph_size;i++)
	printf("Authors: %d\n",auth_num[i]);*/
   

    // free CPU data
    free (newgraph);
    free (auth_num);

    // free the memory allocated on the GPU
    // HERE
    //hipFree(auth_num_gpu);

    return 0;
}

